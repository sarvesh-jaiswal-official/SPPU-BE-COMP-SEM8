#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void matadd(int *l,int *m, int *n)
{
    int x=blockIdx.x;
    int y=blockIdx.y;
    int id=gridDim.x * y +x;
    n[id]=l[id]+m[id];
}
int main()
{
    int a[2][3];
    int b[2][3];
    int c[2][3];
    int *d,*e,*f;
    int i,j;
    printf("\n Enter elements of first matrix of size 2 * 3\n");
    for(i=0;i<2;i++)
    {
        for(j=0;j<3;j++)
            {
                scanf("%d",&a[i][j]);
            }
    }
    printf("\n Enter elements of second matrix of size 2 * 3\n");
        for(i=0;i<2;i++)
        {
            for(j=0;j<3;j++)
                {
                    scanf("%d",&b[i][j]);
                }
        }
    hipMalloc((void **)&d,2*3*sizeof(int));
    hipMalloc((void **)&e,2*3*sizeof(int));
    hipMalloc((void **)&f,2*3*sizeof(int));
 hipMemcpy(d,a,2*3*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(e,b,2*3*sizeof(int),hipMemcpyHostToDevice);
    
dim3 grid(3,2);
/* Here we are defining two dimensional Grid(collection of blocks) structure. Syntax is dim3 grid(no. of columns,no. of rows) */

    matadd<<<grid,1>>>(d,e,f);

 hipMemcpy(c,f,2*3*sizeof(int),hipMemcpyDeviceToHost);
    printf("\nSum of two matrices:\n ");
    for(i=0;i<2;i++)
    {
        for(j=0;j<3;j++)
        {
              printf("%d\t",c[i][j]);
        }
        printf("\n");
    }
    hipFree(d);
    hipFree(e);
    hipFree(f);
    return 0;
}